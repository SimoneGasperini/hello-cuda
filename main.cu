#include "/content/hello-cuda/lib.h"

int main()
{
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	hello_cuda<<<1, 1>>>();
	hipEventRecord(end);

	hipEventSynchronize(end);
	return 0;
}